#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// CUDA function definitions
//
///////////////////////////////////////////////////////////////////////////////
#include "kernel.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

///////////////////////////////////////////////////////////////////////////////
// error check function
#define cuda_safe_call(ans) { simCudaAssert((ans), __FILE__, __LINE__); }
void simCudaAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

///////////////////////////////////////////////////////////////////////////////
// used namespace
using namespace simula;
using namespace simCuda;

///////////////////////////////////////////////////////////////////////////////
// local namespace
namespace {

	/////////////////////////////////////////////////////////////////////////////
	// device variable
	kMolecule* mlist_d;
	kMolecule* mlist_h;

	/////////////////////////////////////////////////////////////////////////////
	// deep copy molecule list to device
	hipError_t mlist_to_dev(simBool free_flag = false)
	{
		// allocate memory
		simSize msize = get_molecule_size();
		mlist_h = (kMolecule*)malloc(msize * sizeof(kMolecule));
		cuda_safe_call(hipMalloc((void**)&mlist_d, msize * sizeof(kMolecule)));
		// deep copy molecule data into C struct
		for (simI1 i = 0; i < msize; ++i) {
			mlist_h[i].x = get_molecule(i).x();
			mlist_h[i].y = get_molecule(i).y();
			mlist_h[i].d = get_molecule(i).d();
			mlist_h[i].i = get_molecule(i).self_id();
			mlist_h[i].t = get_molecule(i).type_id();
		}
		// copy data into device
		hipError_t err = hipMemcpy(mlist_d, mlist_h, msize * sizeof(kMolecule), hipMemcpyHostToDevice);
		// free mlist_h
		if (free_flag) {
			free(mlist_h);
		}
		return err;
	}

	/////////////////////////////////////////////////////////////////////////////
	// deep copy molecule list to host
	hipError_t mlist_to_host()
	{
		// allocate memory
		simSize msize = get_molecule_size();
		// copy data back to host
		hipError_t err = hipMemcpy(mlist_h, mlist_d, msize * sizeof(kMolecule), hipMemcpyDeviceToHost);
		// deep copy back to struct
		if (err = hipSuccess) {
			for (simI1 i = 0; i < msize; ++i) {
				get_molecule(i).set_x(mlist_h[i].x);
				get_molecule(i).set_y(mlist_h[i].y);
				get_molecule(i).set_d(mlist_h[i].d);
			}
		}
		return err;
	}
	/////////////////////////////////////////////////////////////////////////////
	// kernel function
	// ==> to check if its neighboring points are occupied
	__global__ void addKernel(kMolecule* mlist, simI1* r, simI1 size)
	{
		simI1 idx = threadIdx.x;
		simI1 sx = mlist[idx].x, sy = mlist[idx].y;

		r[idx * 4 + 0] = 1; 
		r[idx * 4 + 1] = 1;
		r[idx * 4 + 2] = 1; 
		r[idx * 4 + 3] = 1;
		
		for (simI1 i = 0; i < size; ++i) {
			simI1 tx = mlist[i].x, ty = mlist[i].y;
			if (sx + 1 == tx && sy == ty) { r[idx * 4 + 0] = 0; }
			if (sx == tx && sy + 1 == ty) { r[idx * 4 + 1] = 0; }
			if (sx - 1 == tx && sy == ty) { r[idx * 4 + 2] = 0; }
			if (sx == tx && sy - 1 == ty)	{ r[idx * 4 + 3] = 0; }
		}
		
	}

	simI1* result_d;
	simI1* result_h;

	// Helper function for using CUDA to add vectors in parallel.
	void funcCuda()
	{
		simI1 msize = get_molecule_size();

		// Choose which GPU to run on, change this on a multi-GPU system.
		cuda_safe_call(hipSetDevice(0));

		// Allocate constant memory
		result_h = (simI1*)malloc(4 * msize * sizeof(simI1));
		cuda_safe_call(hipMalloc((void**)&result_d, 4 * msize * sizeof(simI1)));
		
		// Copy data
		cuda_safe_call(mlist_to_dev(true));

		// Launch a kernel on the GPU with one thread for each element.
		addKernel <<< 1, msize >>> (mlist_d, result_d, msize);

		// Check for any errors launching the kernel
		cuda_safe_call(hipGetLastError());

		// Check any errors encountered during the launch.
		cuda_safe_call(hipDeviceSynchronize());

		// Copy output vector from GPU buffer to host memory.
		cuda_safe_call(hipMemcpy(result_h, result_d, 4 * msize * sizeof(simI1), hipMemcpyDeviceToHost));
	}

};

int simCuda::main_temp()
{
	// Test overlap.
	funcCuda();
	// print result
	for (simI1 i = 0; i < get_molecule_size(); ++i) {
		printf("{%d,%d,%d,%d}\n", result_h[4 * i + 0], result_h[4 * i + 1], result_h[4 * i + 2], result_h[4 * i + 3]);
	}

	// for tracing tools such as Nsight and Visual Profiler
	cuda_safe_call(hipDeviceReset());
	return 0;
}