#include "hip/hip_runtime.h"
#include "kernel.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t funcCuda(int *c, const int *a, const int *b, const int* r, unsigned int size);

int main_temp()
{
	const int arraySize = 5;

	const int a[arraySize] = { 1, 2, 3, 4, 5 }; // x position
	const int b[arraySize] = { 1, 1, 1, 1, 1 }; // y position

	int c[arraySize * 4] = { 0 };

	const int r[2] = { 1, 0 };      // constant memory

	// Test overlap.
	hipError_t cudaStatus = funcCuda(c, a, b, r, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "function failed!");
		return 1;
	}

	printf("#1 {1,2,3,4} = {%d,%d,%d,%d}\n", c[0], c[1], c[2], c[3]);
	printf("#2 {1,2,3,4} = {%d,%d,%d,%d}\n", c[4], c[5], c[6], c[7]);
	printf("#3 {1,2,3,4} = {%d,%d,%d,%d}\n", c[8], c[9], c[10], c[11]);
	printf("#4 {1,2,3,4} = {%d,%d,%d,%d}\n", c[12], c[13], c[14], c[15]);
	printf("#5 {1,2,3,4} = {%d,%d,%d,%d}\n", c[16], c[17], c[18], c[19]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// constant memory test
__constant__ int dev_r[2];

// kernel function
__global__ void addKernel(int *c, const int *x, const int *y, const int size)
{
	int idx = threadIdx.x;
	int i = idx % size;
	int j = idx / size;

	int sx = x[i];
	int sy = y[i];
	int tx = x[j];
	int ty = y[j];

	if (sx + 1 == tx && sy == ty) {
		c[i * 4 + 0] = dev_r[0];
	}
	else { 
		c[i * 4 + 0] = dev_r[1];
	}

	if (sx == tx && sy + 1 == ty) {
		c[i * 4 + 1] = dev_r[0];
	}
	else {
		c[i * 4 + 1] = dev_r[1];
	}

	if (sx - 1 == tx && sy == ty) {
		c[i * 4 + 2] = dev_r[0];
	}
	else {
		c[i * 4 + 2] = dev_r[1];
	}

	if (sx == tx && sy - 1 == ty) {
		c[i * 4 + 3] = dev_r[0];
	}
	else {
		c[i * 4 + 3] = dev_r[1];
	}

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t funcCuda(int *c, const int *a, const int *b, const int* r, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	hipError_t cudaStatus;

	printf("r: %d, %d\n", r[0], r[1]);

	try {

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			throw - 1;
		}

		// Allocate constant memory
		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_r), r, 2 * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpyToSymbol failed! Fail to allocate constant memory");
			throw - 1;
		}

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_c, 4 * size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw - 1;
		}

		cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw - 1;
		}

		cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw - 1;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw - 1;
		}

		cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw - 1;
		}

		// Launch a kernel on the GPU with one thread for each element.
		addKernel <<<1, size * size>>>(dev_c, dev_a, dev_b, size);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			throw - 1;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			throw - 1;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, 4 * size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			throw - 1;
		}

	}

	catch (int e) {

		if (e == -1) {
			hipFree(dev_c);
			hipFree(dev_a);
			hipFree(dev_b);
		}
		else {
			throw;
		}

	}

	return cudaStatus;
}
